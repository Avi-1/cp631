#include "hip/hip_runtime.h"
extern "C" { 
    #include "matrix.h"
    #include "basic_functions.h"
    #include "file_io.h"
    #include "convolution.h"
}
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>

__global__ void apply_sobel_convolution_cuda(int *kernel_x, int *kernel_y, int *image, int*output, int kernel_x_dim, int image_height, int image_width){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int index = row * image_width + col;
    if( row <= image_height && col <= image_width){
        int gradient_magnitude, border = kernel_x_dim/2;
        int convolution_sum_x = 0, convolution_sum_y = 0;

        if(row < 1 || col < 1 || row > image_height-border-1 || col > image_width-1){
            output[index] = 255; // Ignore the border
        } else {
            int i, j;
            // Apply x
            for(i = 0-border; i <= border; i++){
                for(j = 0-border; j <= border; j++){
                    convolution_sum_x += kernel_x[(i+border) * kernel_x_dim + j+border] * image[(row+i)*image_width + col+j];
                }
            }

            // Apply y
            for(i = 0-border; i <= border; i++){
                for(j = 0-border; j <= border; j++){
                    convolution_sum_y += kernel_y[(i+border) * kernel_x_dim + j+border] * image[(row+i)*image_width + col+j];
                }
            }

            gradient_magnitude = convolution_sum_x + convolution_sum_y;
            if(gradient_magnitude > 255) gradient_magnitude = 255;
            if(gradient_magnitude < 0) gradient_magnitude = 0;
            output[index] = 255 - gradient_magnitude;
        }
    }
}


int main(int argc, char* argv[]){
    Matrix *kernel_x, *kernel_y, *image;
    char *kern_x, *kern_y, *img, *output;

    kern_x = argv[1];
    kern_y = argv[2];
    img = argv[3];
    output = argv[4];

    kernel_x = get_matrix_from_file(kern_x);
    kernel_y = get_matrix_from_file(kern_y);
    image = get_matrix_from_file(img);

    printf("Image width = %d height= %d. \n", image->width, image->height);
    printf("Kernel X width = %d height= %d. \n", kernel_x->width, kernel_x->height);
    printf("Kernel Y width = %d height= %d. \n", kernel_y->width, kernel_y->height);


    // int temp[image->height][ image->width];
    int shadow[image->height][ image->width];

    int *image_gpu;
    int *convoluted_image_gpu;
    int *kernel_x_gpu;
    int *kernel_y_gpu;

    size_t image_size = image->width * image->height * sizeof(int);
    size_t kernel_size  = kernel_x->width * kernel_x->height * sizeof(int);

    clock_t start = clock();
    hipMalloc((void **) &image_gpu, image_size);
    hipMalloc((void **) &convoluted_image_gpu, image_size);
    hipMalloc((void **) &kernel_x_gpu, kernel_size);
    hipMalloc((void **) &kernel_y_gpu, kernel_size);

    hipMemcpy(image_gpu, image->array, image_size, hipMemcpyHostToDevice);
    //hipMemcpy(convoluted_image_gpu, temp, image_size, hipMemcpyHostToDevice);
    hipMemcpy(kernel_x_gpu, kernel_x->array, kernel_size, hipMemcpyHostToDevice);
    hipMemcpy(kernel_y_gpu, kernel_x->array, kernel_size, hipMemcpyHostToDevice);

    int blocks = 2 * sqrt((image->height * image->width) / 1024);
    dim3 blocksPerGrid(blocks, blocks, 1);
    dim3 threadsPerGrid(32, 32, 1);
    apply_sobel_convolution_cuda<<<blocksPerGrid, threadsPerGrid>>>(kernel_x_gpu, kernel_y_gpu, image_gpu, convoluted_image_gpu, kernel_x->width, image->height, image->width);

    hipMemcpy(shadow, convoluted_image_gpu, image_size, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    if(hipError_t error = hipGetLastError()){
      printf ("Error detected after kernel %d\n", error);
      exit (error);
    }

    float time_spent = (float)(clock() - start) / CLOCKS_PER_SEC;

    write_array_to_file(output, &shadow[0][0], image->height, image->width);

    printf("CUDA code done in %f seconds: Output %s\n", time_spent, output);
    log_timing("cuda", kernel_x->width, time_spent);

    hipFree(image_gpu);
    hipFree(convoluted_image_gpu);
    hipFree(kernel_x_gpu);
    hipFree(kernel_y_gpu);

    free_matrix(kernel_x);
    free_matrix(kernel_y);
    free_matrix(image);

    return 0;
}
