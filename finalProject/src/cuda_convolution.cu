#include "hip/hip_runtime.h"
#include <math.h>
#include<stdlib.h>
#include<stdio.h>
#include"matrix.h"
#include"basic_functions.h"
#include"file_io.h"
#include"convolution.h"

__global__ void apply_sobel_convolution_cuda(int *kernel_x, int *kernel_y, int *image, int*output, int *kernel_x_dim, int *image_dim_x, int *image_dim_y){
    
}

int apply_sobel_convolution (Matrix* kernel_x, Matrix* kernel_y, Matrix* image, int x, int y){
    
    int kernel_sum = 0, border = kernel_x->width / 2;
    int convolution_sum_x = 0, convolution_sum_y = 0;
    int i, j;
        

    if(x < 0 || y < 0 || x == image->width-border || y == image->height-border){
        return 0; // Ignore the border
    }

    // Apply x
    for(i = 0-border; i <= border; i++){
        for(j = 0-border; j <= border; j++){
            convolution_sum_x += get_value(kernel_x, i+border, j+border) * get_value(image, x+i, y+j);
        }
    }

    // Apply y
    for(i = 0-border; i <= border; i++){
        for(j = 0-border; j <= border; j++){
            convolution_sum_y += get_value(kernel_y, i+border, j+border) * get_value(image, x+i, y+j);
        }
    }

    int gradient_magnitude = convolution_sum_x + convolution_sum_y;
    if(gradient_magnitude > 255) gradient_magnitude = 255;
    if(gradient_magnitude < 0) gradient_magnitude = 0;
    return 255 - gradient_magnitude;
}


int main(int argc, char* argv[]){
    Matrix *kernel_x, *kernel_y, *image, *convoluted_matrix;

    kernel_x = get_matrix_from_file("../data/kernel_x.txt");
    kernel_y = get_matrix_from_file("../data/kernel_y.txt");
    image = get_matrix_from_file("../data/imgbw.txt");
    
    convoluted_matrix = create_matrix(image->height, image->width);

    for(int i=0; i<image->height; i++){
        for(int j=0; j<image->width; j++){
            set_value(convoluted_matrix, i, j, apply_sobel_convolution(kernel_x, kernel_y, image, i, j));
        }
    }

    write_matrix_to_file("../data/test_o.txt", convoluted_matrix);
    printf("Done\n");
    return 0;
}
