#include "hip/hip_runtime.h"
extern "C" { 
    #include "matrix.h"
    #include "basic_functions.h"
    #include "file_io.h"
    #include "convolution.h"
}
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

__global__ void apply_sobel_convolution_cuda(int *kernel_x, int *kernel_y, int *image, int*output, int kernel_x_dim, int image_height, int image_width){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int index = row * image_width + col;
    if( row <= image_height && col <= image_width){
        int gradient_magnitude, border = kernel_x_dim/2;
        int convolution_sum_x = 0, convolution_sum_y = 0;

        if(row < 1 || col < 1 || row > image_height-border-1 || col > image_width-1){
            output[index] = 255; // Ignore the border
        } else {
            int i, j;
            // Apply x
            for(i = 0-border; i <= border; i++){
                for(j = 0-border; j <= border; j++){
                    convolution_sum_x += kernel_x[(i+border) * kernel_x_dim + j+border] * image[(row+i)*image_width + col+j];
                }
            }

            // Apply y
            for(i = 0-border; i <= border; i++){
                for(j = 0-border; j <= border; j++){
                    convolution_sum_y += kernel_y[(i+border) * kernel_x_dim + j+border] * image[(row+i)*image_width + col+j];
                }
            }

            gradient_magnitude = convolution_sum_x + convolution_sum_y;
            if(gradient_magnitude > 255) gradient_magnitude = 255;
            if(gradient_magnitude < 0) gradient_magnitude = 0;
            output[index] = 255 - gradient_magnitude;
        }
    }
}

int apply_sobel_convolution (Matrix* kernel_x, Matrix* kernel_y, Matrix* image, int x, int y){
    
    int gradient_magnitude, border = kernel_x->width / 2;
    int convolution_sum_x = 0, convolution_sum_y = 0;
    int i, j;
        

    if(x < 1 || y < 1 || x > image->height-border-1 || y > image->width-border-1){
        return 0; // Ignore the border
    }

    // Apply x
    for(i = 0-border; i <= border; i++){
        for(j = 0-border; j <= border; j++){
            convolution_sum_x += get_value(kernel_x, i+border, j+border) * get_value(image, x+i, y+j);
        }
    }

    // Apply y
    for(i = 0-border; i <= border; i++){
        for(j = 0-border; j <= border; j++){
            convolution_sum_y += get_value(kernel_y, i+border, j+border) * get_value(image, x+i, y+j);
        }
    }
    
    gradient_magnitude = convolution_sum_x + convolution_sum_y;
    if(gradient_magnitude > 255) gradient_magnitude = 255;
    if(gradient_magnitude < 0) gradient_magnitude = 0;
    return 255 - gradient_magnitude;
}

int main(int argc, char* argv[]){
    Matrix *kernel_x, *kernel_y, *image, *convoluted_matrix;

    kernel_x = get_matrix_from_file("../data/kernel_x.txt");
    kernel_y = get_matrix_from_file("../data/kernel_y.txt");
    image = get_matrix_from_file("../data/imgbw.txt");
    convoluted_matrix = create_matrix(image->height, image->width);

    printf("Image width = %d height= %d. \n", image->width, image->height);
    printf("Kernel X width = %d height= %d. \n", kernel_x->width, kernel_x->height);
    printf("Kernel Y width = %d height= %d. \n", kernel_y->width, kernel_y->height);

    int i,j;
    for(i=0; i<image->height; i++){
        for(j=0; j<image->width; j++){
            set_value(convoluted_matrix, i, j, apply_sobel_convolution(kernel_x, kernel_y, image, i, j));
        }
    }

    write_matrix_to_file("../data/test_o.txt", convoluted_matrix);
    printf("Done: Output @ ../data/test_o.txt\n");

    int temp[image->height][ image->width];
    int shadow[image->height][ image->width];

    int *image_gpu;
    int *convoluted_image_gpu;
    int *kernel_x_gpu;
    int *kernel_y_gpu;

    size_t image_size = image->width * image->height * sizeof(int);
    size_t kernel_size  = kernel_x->width * kernel_x->height * sizeof(int);

    hipMalloc((void **) &image_gpu, image_size);
    hipMalloc((void **) &convoluted_image_gpu, image_size);
    hipMalloc((void **) &kernel_x_gpu, kernel_size);
    hipMalloc((void **) &kernel_y_gpu, kernel_size);

    hipMemcpy(image_gpu, image->array, image_size, hipMemcpyHostToDevice);
    hipMemcpy(convoluted_image_gpu, temp, image_size, hipMemcpyHostToDevice);
    hipMemcpy(kernel_x_gpu, kernel_x->array, kernel_size, hipMemcpyHostToDevice);
    hipMemcpy(kernel_y_gpu, kernel_x->array, kernel_size, hipMemcpyHostToDevice);

    int blocks = sqrt((2 * image->height * image->width) / 1024);
    dim3 blocksPerGrid(blocks, blocks, 1);
    dim3 threadsPerGrid(32, 32, 1);
    apply_sobel_convolution_cuda<<<blocksPerGrid, threadsPerGrid>>>(kernel_x_gpu, kernel_y_gpu, image_gpu, convoluted_image_gpu, kernel_x->width, image->height, image->width);

    hipMemcpy(shadow, convoluted_image_gpu, image_size, hipMemcpyDeviceToHost);


    hipDeviceSynchronize();
    if(hipError_t error = hipGetLastError()){
      printf ("Error detected after kernel %d\n", error);
      exit (error);
    }

    write_array_to_file("../data/test_gpu.txt", &shadow[0][0], image->height, image->width);

    printf("GPU Done: Output @ ../data/test_gpu.txt\n");

    return 0;
}
